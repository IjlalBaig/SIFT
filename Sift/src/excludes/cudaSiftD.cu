#include "hip/hip_runtime.h"
#include "cudaSiftD.h"
#include <stdio.h>

__global__ void subtractKernel( float *gDst, float *gSrc1, float *gSrc2
							, int w, int p, int h )
{
	int gx = threadIdx.x + blockDim.x * blockIdx.x;
	int gy = threadIdx.y + blockDim.y * blockIdx.y;
	int gIdx = gx + p * gy;

	// Compute difference
	if (gx < w && gy < h)
		gDst[gIdx] = gSrc1[gIdx] - gSrc2[gIdx];
}

__global__ void hessianKernel( float *gDst, float *gSrc
							, int w, int p, int h
							, const int nTilesX, const int nTilesY
							, const int apronLeft, const int apronRight, const int apronUp, const int apronDown
							, const int bankOffset )
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int sx = 0;
	int sy = 0;
	int bDimX = blockDim.x;
	int bDimY = blockDim.y;
	int bIdxX = blockIdx.x;
	int bIdxY = blockIdx.y;
	int gx = tx + bDimX * bIdxX * nTilesX;
	int gy = ty + bDimY * bIdxY * nTilesY;
	int gx_ = 0;
	int dataSizeX = nTilesX*bDimX;
	int sDimX = (apronLeft + dataSizeX + apronRight + bankOffset);
	int gDim = p*h;
	int Dxx = 0;
	int Dxy = 0;
	int Dyy = 0;
	extern __shared__ float shared[];

	// Load data to shared
	cudaMemcpyGlobalToShared( shared, gSrc, tx, ty
							, gx, gy, bDimX, bDimY, w, p, h
							, nTilesX, nTilesY
							, apronLeft, apronRight, apronUp, apronDown, bankOffset );

	for (int i = 0; i < nTilesX; ++i)
	{
		gx_ = gx + i*bDimX;
		if (gx_ < w && gy < h)
		{
			sx = apronLeft + tx + i*bDimX;
			sy = apronUp + ty;

			//	Compute Dxx
			Dxx = -2*shared[cuda2DTo1D( sx, sy, sDimX )];
			Dxx += shared[cuda2DTo1D( sx - 1, sy, sDimX )];
			Dxx += shared[cuda2DTo1D( sx + 1, sy, sDimX )];

			//	Compute Dyy
			Dyy = -2*shared[cuda2DTo1D( sx, sy, sDimX )];
			Dyy += shared[cuda2DTo1D( sx, sy - 1, sDimX )];
			Dyy += shared[cuda2DTo1D( sx, sy + 1, sDimX )];

			//	Compute Dxy
			Dxy = shared[cuda2DTo1D( sx, sy, sDimX )];
			Dxy += shared[cuda2DTo1D( sx - 1, sy - 1, sDimX )];
			Dxy -= shared[cuda2DTo1D( sx - 1, sy, sDimX )];
			Dxy -= shared[cuda2DTo1D( sx, sy - 1, sDimX )];

			// Copy data to global
			gDst[cuda2DTo1D( gx_, gy, p )] = Dxx;
			gDst[cuda2DTo1D( gx_, gy, p ) + gDim] = Dyy;
			gDst[cuda2DTo1D( gx_, gy, p ) + 2*gDim] = Dxy;
		}
	}

}

__global__ void xblurMultiKernel( float *gDst, float *gSrc
								, int w, int p, int h
								, const int nTilesX, const int nTilesY
								, const int apronLeft, const int apronRight, const int apronUp, const int apronDown
								, const int bankOffset)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bDimX = blockDim.x;
	int bDimY = blockDim.y;
	int bIdxX = blockIdx.x;
	int bIdxY = blockIdx.y;
	int sx = 0;
	int sy = 0;
	int gx = tx + bDimX * bIdxX * nTilesX;
	int gx_ = 0;
	int gy = ty + bDimY * bIdxY * nTilesY;
	int dataSizeX = nTilesX*bDimX;
	int sDimX = (apronLeft + dataSizeX + apronRight + bankOffset);
	int gDim = p*h;
	extern __shared__ float shared[];

	// Load data to shared
	cudaMemcpyGlobalToShared( shared, gSrc, tx, ty
							, gx, gy, bDimX, bDimY, w, p, h
							, nTilesX, nTilesY
							, apronLeft, apronRight, apronUp, apronDown, bankOffset );

	// Convolve-x
	for (int i = 0; i < N_SCALES + 3; ++i)
	{
		int kernelStartIdx = i * B_KERNEL_SIZE;
		for (int j = 0; j < nTilesX; ++j)
		{
			sx = tx + j*bDimX;
			sy = ty;
			gx_ = sx + bDimX * bIdxX * nTilesX;

			if (sx < dataSizeX && gx_ < w && gy < h)
			{
				float sum = 0;
				for (int k = 0; k < B_KERNEL_SIZE; ++k)
					sum = __fmaf_rn( c_GaussianBlur[kernelStartIdx + k], shared[cuda2DTo1D( sx + k, sy, sDimX )], sum );
				__syncthreads();
				gDst[cuda2DTo1D( gx + j*bDimX, gy, p ) + i*gDim] = sum;
			}
		}
//		// Copy data to global
//		cudaMemcpySharedToGlobal(gDst, shared
//								, tx, ty, gx, gy
//								, bDimX, bDimY, w, p, h
//								, nTilesX, nTilesY
//								, apronLeft, apronRight, apronUp, apronDown, bankOffset);
	}
}

__global__ void yblurKernel( float *gDst, float *gSrc, const int scaleIdx
							, int w, int p, int h
							, const int nTilesX, const int nTilesY
							, const int apronLeft, const int apronRight, const int apronUp, const int apronDown
							, const int bankOffset)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bDimX = blockDim.x;
	int bDimY = blockDim.y;
	int bIdxX = blockIdx.x;
	int bIdxY = blockIdx.y;
	int sx = 0;
	int sy = 0;
	int gx = tx + bDimX * bIdxX * nTilesX;
	int gy = ty + bDimY * bIdxY * nTilesY;
	int gy_ = 0;
	int dataSizeY = nTilesY*bDimY;
	int sDimX = (bDimX + bankOffset);
	int kernelStartIdx = scaleIdx * B_KERNEL_SIZE;
	extern __shared__ float shared[];

	// Load data to shared
	cudaMemcpyGlobalToShared( shared, gSrc, tx, ty
							, gx, gy, bDimX, bDimY, w, p, h
							, nTilesX, nTilesY
							, apronLeft, apronRight, apronUp, apronDown, bankOffset );

	// Convolve-y
	for (int i = 0; i < nTilesY; ++i)
	{
		sx = tx;
		sy = ty + i*bDimY;
		gy_ = sy + bDimY * bIdxY * nTilesY;

		if (sy < dataSizeY && gy_ < h && gx < w)
		{
				float sum = 0;
				for (int j = 0; j < B_KERNEL_SIZE; ++j)
					sum = __fmaf_rn( c_GaussianBlur[kernelStartIdx + j], shared[cuda2DTo1D(sx, sy + j, sDimX)], sum );
				__syncthreads();
				gDst[cuda2DTo1D( gx, gy + i*bDimY, p )] = sum;
		}
	}

}

__global__ void copyKernel( float *gDst, const float *gSrc, int w, int p, int h )
{
	int gx = threadIdx.x + blockDim.x * blockIdx.x;
	int gy = threadIdx.y + blockDim.y * blockIdx.y;
	int gIdx = gx + p * gy;
	gDst[gIdx] = gSrc[gIdx];
}

__global__ void subtractKernel( float *gDst, const float *gSrc1, const float *gSrc2, int w, int p, int h )
{
	int gx = threadIdx.x + blockDim.x * blockIdx.x;
	int gy = threadIdx.y + blockDim.y * blockIdx.y;
	int gIdx = gx + p * gy;

	// Compute difference
	if (gx < w && gy < h)
		gDst[gIdx] = gSrc1[gIdx] - gSrc2[gIdx];
}

__global__ void resizeKernel( float *gDst, float *gSrc, int w, int p, int h )
{
	int gx = threadIdx.x + blockDim.x * blockIdx.x;
	int gy = threadIdx.y + blockDim.y * blockIdx.y;
	int gIdx = gx + p * gy;

	// Resize to half size
	if(gx < w && gy < h && gx%2 == 0 && gy%2 == 0 )
	{
		int gx_ = gx / 2.0f;
		int gy_ = gy / 2.0f;
		int p_ = cudaIAlignUp( cudaIDivUpNear( w, 2 ), 128 );
		int gIdx_ = gx_ + gy_ * p_;
		gDst[gIdx_] = gSrc[gIdx];
	}
}


__global__ void kernelGaussianSize()
{
	int tx = threadIdx.x;
	printf( "scale %d\t:\t%d\n", tx, c_GaussianBlurSize[tx] );
	if (tx == 0)
		printf( "%d\n", c_MaxGaussianBlurSize );
}

__global__ void kernelGaussianVector()
{
	int tx = threadIdx.x;
	printf( "thread %d\t:\t%f\n", tx, c_GaussianBlur[tx] );
}

__global__ void kernel()
{
	int count = 0;
	for (int i = 0; i < 5000; ++i)
		count++;
	printf( "hi this is thread:%d\t%d\n", threadIdx.x, count );
}

__global__ void shKernel( float *data
						, int w, int p, int h
						, const int nTilesX, const int nTilesY
						, const int apronLeft, const int apronRight, const int apronUp, const int apronDown )
{
	extern __shared__ float shared[];
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bDimX = blockDim.x;
	int bDimY = blockDim.y;
	int bIdxX = blockIdx.x;
	int bIdxY = blockIdx.y;
	int gx = tx + bDimX * bIdxX;
	int gy = ty + bDimY * bIdxY;
//	if (gx == 0 && gy == 0)
//	{
//		for (int j = 0; j < h; ++j)
//		{
//			for (int i = 0; i < p; ++i)
//			{
//				printf( "%f  ", data[cuda2DTo1D( i, j, p )] );
//			}
//			printf( "\n" );
//		}
//
//	}
	cudaMemcpyGlobalToShared( shared, data
							, tx, ty, gx, gy
							, bDimX, bDimY, w, p, h
							, nTilesX, nTilesY
							, apronLeft, apronRight, apronUp, apronDown, 0 );

	__syncthreads();
	if (gx == 0 && gy == 0)
	{
		for (int j = 0; j < apronUp +  bDimY*nTilesY + apronDown; ++j)
		{
			for (int i = 0; i < apronLeft + bDimX*nTilesX + apronRight; ++i)
			{
				printf( "%f  ", shared[cuda2DTo1D( i, j, apronLeft + bDimX*nTilesX + apronRight )] );
			}
			printf( "\n" );
		}
	}

	cudaMemcpySharedToGlobal( data, shared
							, tx, ty, gx, gy
							, bDimX, bDimY, w, p, h
							, nTilesX, nTilesY
							, apronLeft, apronRight, apronUp, apronDown, 0 );
//	if (gx == 0 && gy == 0)
//	{
//		for (int j = 0; j < h; ++j)
//		{
//			for (int i = 0; i < w; ++i)
//			{
//				printf( "%f  ", data[cuda2DTo1D( i, j, p)] );
//			}
//			printf( "\n" );
//		}
//
//	}
}

#include "hip/hip_runtime.h"
#include "excludes/cudaSiftD.cu"
#include <stdio.h>
#include "utils.h"
#include "cudaUtils.h"



void blurOctave(float *dst, float *src, int width, int pitch, int height, hipStream_t &stream)
{
	float k = pow(2,0.5);
	// Get max Kernel size
	int maxKernelSize = imfilter::gaussianSize( pow( k, N_SCALES + 1 ) * SIGMA );
	// Get max apron Size
	int maxApronStart = floor( maxKernelSize / 2 );
	int maxApronEnd =  maxKernelSize - maxApronStart - 1;
	// Set bankOffset to 1 for even filter size
	int bankOffset = (maxApronStart == maxApronEnd) ? (1):(0);
	// Set x-convolution kernel parameters
	int nTilesX = 11;
	int nTilesY = 1;
	dim3 blockSize(WIDTH_CONV_BLOCK, HEIGHT_CONV_BLOCK, 1);
	dim3 gridSize(iDivUp(width, WIDTH_CONV_BLOCK*nTilesX), iDivUp(height, HEIGHT_CONV_BLOCK*nTilesY), 1);
	int sDimX = nTilesX*WIDTH_CONV_BLOCK + maxApronStart + maxApronEnd + bankOffset;
	int sDimY = HEIGHT_CONV_BLOCK;

	printf("sDimX \t%d\nsDimY \t%d\n", sDimX, sDimY);
	blurKernel<<<gridSize, blockSize, sDimX*sDimY*sizeof(float), stream>>>(dst, src
																		, width, pitch, height
																		, nTilesX, nTilesY
																		, maxApronStart, maxApronEnd, 0, 0
																		, bankOffset);
}
















void initDeviceConstant()
{


	// Set c_GaussianBlurSize[] for each scale
	// Set c_MaxGaussianBlurSize
	// Set c_GaussianBlur[] kernel for each scale
	float k = pow(2,0.5);
	int blurSize = 0;
	int maxBlurSize = 0;
	int blurSizeArray[N_SCALES + 3];

	int kernelStartPtr = 0;
	float gaussianBlur[B_KERNEL_SIZE];
	float sigma = 0.0;
	float sigmaOld = 0.0;
	float sigmaNew = 0.0;

	for(int i = 0; i < N_SCALES + 3; ++i)
	{
		sigma = pow( k, i-1 ) * SIGMA;
		sigmaNew  = sigma - sigmaOld;
		sigmaOld = sigma;
		// Push new kernel array to gaussiaBlur[]
		imfilter::gaussian1D( gaussianBlur + kernelStartPtr, sigmaNew );
		// Set blurSize to current kernel size
		blurSize = imfilter::gaussianSize( sigmaNew );
		// Push blurSize to blurSizeArray[]
		blurSizeArray[i] = blurSize;
		// Increment kernelStartPtr to point on top of gaussiaBlur[] stack
		kernelStartPtr += blurSize;
		// Set maxBlurSize
		maxBlurSize = (blurSize > maxBlurSize) ? (blurSize):(maxBlurSize);
	}
	// Copy symbols to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL( c_GaussianBlurSize), &blurSizeArray, (N_SCALES + 3)*sizeof( int ) );
	hipMemcpyToSymbol(HIP_SYMBOL( c_MaxGaussianBlurSize), &maxBlurSize, sizeof( int ) );
	hipMemcpyToSymbol(HIP_SYMBOL( c_GaussianBlur), &gaussianBlur, B_KERNEL_SIZE * sizeof( float ) );
}

void testSetConstants(hipStream_t &stream)
{
	kernelGaussianSize<<<1, 5, 0, stream>>>();
	kernelGaussianVector<<<1, B_KERNEL_SIZE, 0, stream>>>();
}

void testcopyKernel( hipStream_t &stream )
{
	kernel<<<1, 10, 0, stream>>>();
}

void sharedKernel( hipStream_t &stream )
{
	float *d_data;
	float *h_data;
	int w = 9;
	int p = 10;
	int h = 5;
	int gx = 50;
	h_data = (float *)malloc( gx * sizeof( float));
	CUDA_SAFECALL( hipMalloc( (void **)&d_data, (size_t)gx * sizeof( float) ) );

	for (int i = 0; i < p; ++i)
	{
		for (int j = 0; j < h; ++j)
			h_data[i + j*p] = (i < w) ? ((i+1)*(i+1)): -1;
	}

	CUDA_SAFECALL( hipMemcpy((void *)d_data, (void *)h_data, (size_t)(gx * sizeof(float)), hipMemcpyHostToDevice ) );
	int nTilesX = 2;
	int nTilesY = 1;

	int apronLeft = 2;
	int apronRight = 5;
	int apronUp = 2;
	int apronDown = 5;
	dim3 blockDim(3,2,1);
	dim3 gridDim(1,1,1);
	int sx = apronLeft + apronRight + blockDim.x*nTilesX;
	int sy = apronUp + apronDown + blockDim.y*nTilesY;
	shKernel<<<gridDim, blockDim, sx*sy*sizeof( float ), stream>>>( d_data
																	, w, p, h
																	, nTilesX, nTilesY
																	, apronLeft, apronRight, apronUp, apronDown);

	free( h_data );
	CUDA_SAFECALL( hipFree( d_data ));
}

#include "hip/hip_runtime.h"
#include "excludes/cudaSiftD.cu"
#include <stdio.h>
#include "utils.h"
#include "cudaUtils.h"



void blurOctave(float *dst, float *src, int width, int pitch, int height, hipStream_t &stream)
{
	float k = pow(2,0.5);

	// Get max apron Size
	int maxApronStart = B_KERNEL_RADIUS;
	int maxApronEnd = B_KERNEL_RADIUS;
	// Set bankOffset to 1 for even filter size
	int bankOffset = 0;
	// Set x-convolution kernel parameters
	int nTilesX = 12 - iDivUp((maxApronStart + maxApronEnd), WIDTH_CONV_BLOCK) ;
	int nTilesY = 1;
	dim3 blockSize(WIDTH_CONV_BLOCK, HEIGHT_CONV_BLOCK, 1);
	dim3 gridSize(iDivUp(width, WIDTH_CONV_BLOCK*nTilesX), iDivUp(height, HEIGHT_CONV_BLOCK*nTilesY), 1);
	int sDimX = nTilesX*WIDTH_CONV_BLOCK + maxApronStart + maxApronEnd + bankOffset;
	int sDimY = HEIGHT_CONV_BLOCK;
	blurKernel<<<gridSize, blockSize, sDimX*sDimY*sizeof(float), stream>>>(dst, src
																		, width, pitch, height
																		, nTilesX, nTilesY
																		, maxApronStart, maxApronEnd, 0, 0
																		, bankOffset);
}
















void initDeviceConstant()
{
	// Set c_GaussianBlur[] kernel for each scale
	float k = pow(2,0.5);
	float gaussianBlur[(N_SCALES + 3) * B_KERNEL_SIZE];
	float sigmaNew = 0.0;

	for(int i = 0; i < N_SCALES + 3; ++i)
	{
		sigmaNew = pow( k, i-1 ) * SIGMA;

		// Push new kernel array to gaussiaBlur[]
		imfilter::gaussian1D( gaussianBlur + i * (2*B_KERNEL_RADIUS + 1), sigmaNew );
	}
	// Copy symbols to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL( c_GaussianBlur), &gaussianBlur, (N_SCALES + 3) * B_KERNEL_SIZE * sizeof( float ) );
}

void testSetConstants(hipStream_t &stream)
{
	kernelGaussianSize<<<1, 5, 0, stream>>>();
	kernelGaussianVector<<<1, (N_SCALES + 3) * B_KERNEL_SIZE, 0, stream>>>();
}

void testcopyKernel( hipStream_t &stream )
{
	kernel<<<1, 10, 0, stream>>>();
}

void sharedKernel( hipStream_t &stream )
{
	float *d_data;
	float *h_data;
	int w = 9;
	int p = 10;
	int h = 5;
	int gx = 50;
	h_data = (float *)malloc( gx * sizeof( float));
	CUDA_SAFECALL( hipMalloc( (void **)&d_data, (size_t)gx * sizeof( float) ) );

	for (int i = 0; i < p; ++i)
	{
		for (int j = 0; j < h; ++j)
			h_data[i + j*p] = (i < w) ? ((i+1)*(i+1)): -1;
	}

	CUDA_SAFECALL( hipMemcpy((void *)d_data, (void *)h_data, (size_t)(gx * sizeof(float)), hipMemcpyHostToDevice ) );
	int nTilesX = 2;
	int nTilesY = 1;

	int apronLeft = 2;
	int apronRight = 5;
	int apronUp = 2;
	int apronDown = 5;
	dim3 blockDim(3,2,1);
	dim3 gridDim(1,1,1);
	int sx = apronLeft + apronRight + blockDim.x*nTilesX;
	int sy = apronUp + apronDown + blockDim.y*nTilesY;
	shKernel<<<gridDim, blockDim, sx*sy*sizeof( float ), stream>>>( d_data
																	, w, p, h
																	, nTilesX, nTilesY
																	, apronLeft, apronRight, apronUp, apronDown);

	free( h_data );
	CUDA_SAFECALL( hipFree( d_data ));
}

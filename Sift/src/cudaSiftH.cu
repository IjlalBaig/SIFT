#include "excludes/cudaSiftD.cu"
#include <stdio.h>
#include "cudaUtils.h"

void testcopyKernel( hipStream_t &stream )
{
	kernel<<<1, 10, 0, stream>>>();
}

void sharedKernel( hipStream_t &stream )
{
	float *d_data;
	float *h_data;
	int w = 9;
	int p = 10;
	int h = 5;
	int gx = 50;
	h_data = (float *)malloc( gx * sizeof( float));
	CUDA_SAFECALL( hipMalloc( (void **)&d_data, (size_t)gx * sizeof( float) ) );

	for (int i = 0; i < p; ++i)
	{
		for (int j = 0; j < h; ++j)
			h_data[i + j*p] = (i < w) ? (i*i): -1;
	}

	CUDA_SAFECALL( hipMemcpy((void *)d_data, (void *)h_data, (size_t)(gx * sizeof(float)), hipMemcpyHostToDevice ) );

	int sx = 5+4; // if size is incorrect shared memory will still work fine
	int sy = 2+8;
	dim3 blockDim(4,2,1);
	dim3 gridDim(1,1,1);
	shKernel<<<gridDim, blockDim, sx*sy*sizeof( float ), stream>>>( d_data, w, p, h, 5, 0, 3, 5 );

	free( h_data );
	CUDA_SAFECALL( hipFree( d_data ));
}

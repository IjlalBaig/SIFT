#include "hip/hip_runtime.h"
#include "sift.h"
#include "utils.h"
#include "cudaUtils.h"
#include "cudaSiftH.h"
#include "cudaImage.h"


int sift( std::string dstPath, std::string *srcPath)
{
//	hipStream_t stream;
//	CUDA_SAFECALL( hipStreamCreate(&stream) );
//	sharedKernel( stream );
//	CUDA_SAFECALL( hipStreamDestroy(stream));

	//	Load image batch to Mat object
	cv::Mat matImg[BATCH_SIZE];
	int width[BATCH_SIZE];
	int height[BATCH_SIZE];
	int pitch[BATCH_SIZE];
	for (int i = 0; i < BATCH_SIZE; ++i)
	{
		image::imload( matImg[i], srcPath[i], false );
		width[i] = matImg[i].cols;
		height[i] = matImg[i].rows;
	}
	/*
	 *
	 * clean */
	cv::Mat1f matRes(matImg[0].rows, matImg[0].cols);
	CudaImage cuRes;
	cuRes.Allocate( matImg[0].cols, matImg[0].rows, NULL, (float *)matRes.data );
	/********/
	//	Allocate Cuda Objects
	CudaImage cuImg[BATCH_SIZE];
	SiftData siftData[BATCH_SIZE];
	for (int i = 0; i < BATCH_SIZE; ++i)
	{
		cuImg[i].Allocate( width[i], height[i], NULL, (float *)matImg[i].data );
		siftData[i].Allocate(MAX_POINTS, NULL, NULL);
		pitch[i] = cuImg[i].pitch;
	}

	//	Create batch streams
	hipStream_t stream[BATCH_SIZE];
	for (int i = 0; i < BATCH_SIZE; ++i)
		CUDA_SAFECALL( hipStreamCreate( &stream[i] ) );

	// 	Set device constants
		initDeviceConstant();

	//	Execute sift on streams
	for (int i = 0; i < BATCH_SIZE; ++i)
	{
		//	Upload CudaImage to GPU
		cuImg[i].Upload(stream[i]);
		siftData[i].Upload(stream[i]);
		/*
		 *
		 * clean */
		cuRes.Upload(stream[0]);
		/*********/
		//	Launch Kernels
//		testcopyKernel(stream[i]);
//		testSetConstants(stream[i]);

		//	Allocate octave pointers
		float *(d_srcImage[i]);
		float *(d_multiBlur[i]);
		float *(d_multiDoG[i]);
		float *(d_multiHessian[i]);
		float *(d_multiMagnitude[i]);
		float *(d_multiDirection[i]);

		for (int j = 0; j < N_OCTAVES; ++j)
		{

			allocateOctave( d_multiBlur[i], d_multiDoG[i]
			            , d_multiHessian[i], d_multiMagnitude[i], d_multiDirection[i]
						, width[i], pitch[i], height[i] );
			//	Compute octave scale space
			blurOctave( d_multiBlur[i], cuImg[i].d_data, width[i], pitch[i], height[i], stream[i] );

			//	Copy data to result image
			int gDim = cuRes.pitch*cuRes.height;
			copyDeviceData(cuRes.d_data, 4*gDim + d_multiBlur[i], cuRes.width, cuRes.pitch, cuRes.height, stream[i] );
			//	Free octave pointers
			freeOctave(d_multiBlur[i], d_multiDoG[i]
					, d_multiHessian[i], d_multiMagnitude[i], d_multiDirection[i] );
		}

	}

	for (int i = 0; i < BATCH_SIZE; ++i)
	{
		//	Download results to CPU
		cuImg[i].Readback(stream[i]);
		siftData[i].Readback(stream[i]);
		/*
		 *
		 * clean */
		cuRes.Readback(stream[i]);
		/**********/
	}

	//	Show result
	image::imshow( matRes );
//	image::imshow( matImg[0] );

	//	Destroy cuda streams for batchSize
	for (int i = 0; i < BATCH_SIZE; ++i)
		CUDA_SAFECALL( hipStreamDestroy(stream[i]));
	return 0;
}

SiftData::SiftData():
	numPts( 0 ), maxPts( 0 ), h_data( NULL ), d_data( NULL ),
	d_internalAlloc( false ), h_internalAlloc( false )
{

}
SiftData::~SiftData()
{
	if(d_internalAlloc && d_data!=NULL)
		CUDA_SAFECALL( hipFree( d_data ) );
	d_data = NULL;
	if(h_internalAlloc && h_data!=NULL)
		free( h_data );
	h_data = NULL;
}

void SiftData::Allocate(int max, SiftPoint *d_ptr, SiftPoint *h_ptr)
{
	numPts = 0;
	maxPts = max;
	d_data = d_ptr;
	h_data = h_ptr;
	if (d_ptr==NULL)
	{
		CUDA_SAFECALL( hipMalloc( (void **)&d_data, (size_t)(maxPts * sizeof( SiftPoint )) ) );
		if (d_data==NULL)
			printf( "Failed to allocate Sift device data\n" );
		d_internalAlloc = true;
	}
	if (h_ptr==NULL)
	{
		h_data = (SiftPoint *)malloc( maxPts * sizeof( SiftPoint ) );
		if (h_data==NULL)
			printf( "Failed to allocate Sift host data\n" );
		h_internalAlloc = true;
	}
}

double SiftData::Upload(hipStream_t stream)
{
	if (d_data!=NULL && h_data!=NULL)
		CUDA_SAFECALL( hipMemcpyAsync( (void *)d_data, (const void *)h_data, maxPts * sizeof( SiftPoint ), hipMemcpyHostToDevice, stream ) );
	return 0.0;
}

double SiftData::Readback(hipStream_t stream)
{
	CUDA_SAFECALL( hipMemcpyAsync( (void *)h_data, (const void *)d_data, maxPts * sizeof( SiftPoint ), hipMemcpyDeviceToHost, stream ) );
	return 0.0;
}
